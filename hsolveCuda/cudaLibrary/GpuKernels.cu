#include "hip/hip_runtime.h"
/**********************************************************************
** This program is part of 'MOOSE', the
** Messaging Object Oriented Simulation Environment.
**   copyright (C) 2003-2007 Upinder S. Bhalla, Niraj Dudani and NCBS
** It is made available under the terms of the
** GNU Lesser General Public License version 2.1
** See the file COPYING.LIB for the full notice.
**********************************************************************/

#include <cstdio>
#include "GpuKernels.h"
#include "../HSolveStruct.h"	// For CompartmentStruct, etc.
#include "../HinesMatrix.h"		// For JunctionStruct
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void updateMatrixKernel(GpuDataStruct ds) {
	/*
	 * Copy contents of HJCopy_ into HJ_. Cannot do a vector assign() because
	 * iterators to HJ_ get invalidated in MS VC++
	 */
	if ( ds.HJSize != 0 )
		memcpy( ds.HJ, ds.HJCopy, sizeof( double ) * ds.HJSize );

	double *ihs = ds.HS;
	double *iv  = ds.V;
	
	CompartmentStruct *ic;
	for ( 	ic = ds.compartment;
			ic < ds.compartment + ds.nCompts * sizeof( CompartmentStruct );
			++ic )
	{
		*ihs         = *( 2 + ihs );
		*( 3 + ihs ) = *iv * ic->CmByDt + ic->EmByRm;
		
		ihs += 4, ++iv;
	}
	
	/* Not going to consider inject at the first implementation level
	map< unsigned int, InjectStruct >::iterator inject;
	for ( inject = inject_.begin(); inject != inject_.end(); inject++ ) {
		unsigned int ic = inject->first;
		InjectStruct& value = inject->second;
		
		HS_[ 4 * ic + 3 ] += value.injectVarying + value.injectBasal;
		
		value.injectVarying = 0.0;
	}
	*/
}

__global__ void forwardEliminateKernel(GpuDataStruct ds) {
	unsigned int ic = 0;
	double *ihs = ds.HS;
	double **iop = ds.operand;
	JunctionStruct *junction;
	
	if ( iop ) {
		for( int x = 0 ; x < 36 ; x++ ) {		//XXX debugging only
			printf( "%p ", *(iop + x) );
		}
		printf("\n");
	}
	
	double pivot;
	double division;
	unsigned int index;
	unsigned int rank;
	double *j, *s;
	for ( junction = ds.junction;
	      junction < ds.junction + ds.junctionSize;
	      junction++ )
	{
		index = junction->index;
		rank = junction->rank;
		
		while ( ic < index ) {
			*( ihs + 4 ) -= *( ihs + 1 ) / *ihs * *( ihs + 1 );
			*( ihs + 7 ) -= *( ihs + 1 ) / *ihs * *( ihs + 3 );
			
			++ic, ihs += 4;
		}
		
		pivot = *ihs;
		if ( rank == 1 ) {
			printf("rank=1; ");
			printf("ic: %d ", ic);
			printf("ihs: %p ", ihs);
			printf("iop: %p ", iop);
			j = *iop;
			s = *(iop + 1);
			
			printf( "s: %p\n", s );
			
			division    = *( j + 1 ) / pivot;
			*( s )     -= division * *j;
			*( s + 3 ) -= division * *( ihs + 3 );
			
			iop += 3;
		} else if ( rank == 2 ) {
			printf("rank=2; ");
			printf("ic: %d ", ic);
			printf("ihs: %p ", ihs);
			printf("iop: %p ", iop);
			j = *iop;
			
			s           = *( iop + 1 );
			printf( "s: %p ", s );
			division    = *( j + 1 ) / pivot;
			*( s )     -= division * *j;
			*( j + 4 ) -= division * *( j + 2 );
			*( s + 3 ) -= division * *( ihs + 3 );
			
			s           = *( iop + 3 );
			printf( "s: %p\n", s );
			division    = *( j + 3 ) / pivot;
			*( j + 5 ) -= division * *j;
			*( s )     -= division * *( j + 2 );
			*( s + 3 ) -= division * *( ihs + 3 );
			
			iop += 5;
		} else {
			printf("rank=%d; ", rank);
			printf("ic: %d ", ic);
			printf("ihs: %p ", ihs);
			printf("iop: %p\n", iop);
			double **end = iop + 3 * rank * ( rank + 1 );
			for ( ; iop < end; iop += 3 )
				**iop -= **( iop + 2 ) / pivot * **( iop + 1 );
		}
		
		++ic, ihs += 4;
	}
	
	while ( ic < ds.nCompts - 1 ) {
		*( ihs + 4 ) -= *( ihs + 1 ) / *ihs * *( ihs + 1 );
		*( ihs + 7 ) -= *( ihs + 1 ) / *ihs * *( ihs + 3 );
		
		++ic, ihs += 4;
	}
}

__global__ void backwardSubstituteKernel(GpuDataStruct ds) {
	// We are reverse iterating here, so all pointers are initialized to the
	// ultimate elements of their respective arrays.
	int ic = ds.nCompts - 1;
	double *ivmid = ds.VMid + ic;
	double *iv = ds.V + ic;
	double *ihs = ds.HS + 4 * ds.nCompts - 1;
	double **iop = ds.operand + ds.operandSize - 1;
	double **ibop = ds.backOperand + ds.backOperandSize - 1;
	JunctionStruct *junction = ds.junction + ds.junctionSize - 1;
	
	*ivmid = *ihs / *( ihs - 3 );
	*iv = 2 * *ivmid - *iv;
	--ic, --ivmid, --iv, ihs -= 4;
	
	int index;
	int rank;
	for ( ;
	      ds.junction != NULL && junction >= ds.junction;
	      junction-- )
	{
		index = junction->index;
		rank = junction->rank;
		
		while ( ic > index ) {
			// ivmid was -1, so now it's +1!
			*ivmid = ( *ihs - *( ihs - 2 ) * *( ivmid + 1 ) ) / *( ihs - 3 );
			*iv = 2 * *ivmid - *iv;
			
			--ic, --ivmid, --iv, ihs -= 4;
		}
		
		if ( rank == 1 ) {
			*ivmid = ( *ihs - **iop * **( iop - 2 ) ) / *( ihs - 3 );
			
			iop -= 3;
		} else if ( rank == 2 ) {
			double *v0 = *( iop );
			double *v1 = *( iop - 2 );
			double *j  = *( iop - 4 );
			
			*ivmid = ( *ihs
			           - *v0 * *( j + 2 )	// j was a vdIterator in forward!
			           - *v1 * *j			// so + remains +!!
			         ) / *( ihs - 3 );
			
			iop -= 5;
		} else {
			*ivmid = *ihs;
			for ( int i = 0; i < rank; ++i ) {
				*ivmid -= **ibop * **( ibop - 1 );
				ibop -= 2;
			}
			*ivmid /= *( ihs - 3 );
			
			iop -= 3 * rank * ( rank + 1 );
		}
		
		*iv = 2 * *ivmid - *iv;
		--ic, --ivmid, --iv, ihs -= 4;
	}
	
	while ( ic >= 0 ) {
		// The ivmid was -1, so now it becomes +1!
		*ivmid = ( *ihs - *( ihs - 2 ) * *( ivmid + 1 ) ) / *( ihs - 3 );
		*iv = 2 * *ivmid - *iv;
		
		--ic, --ivmid, --iv, ihs -= 4;
	}
}

