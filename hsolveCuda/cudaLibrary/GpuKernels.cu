#include "hip/hip_runtime.h"
/**********************************************************************
** This program is part of 'MOOSE', the
** Messaging Object Oriented Simulation Environment.
**   copyright (C) 2003-2007 Upinder S. Bhalla, Niraj Dudani and NCBS
** It is made available under the terms of the
** GNU Lesser General Public License version 2.1
** See the file COPYING.LIB for the full notice.
**********************************************************************/

#include "GpuKernels.h"
#include "../HsolveStruct.h"	// For CompartmentStruct, etc.
#include "../HinesMatrix.h"		// For JunctionStruct

__global__ void updateMatrixKernel(GpuDataStruct ds) {
	/*
	 * Copy contents of HJCopy_ into HJ_. Cannot do a vector assign() because
	 * iterators to HJ_ get invalidated in MS VC++
	 */
	if ( ds.HJSize != 0 )
		memcpy( ds.HJ, ds.HJCopy, sizeof( double ) * ds.HJSize );
	
	double *ihs = ds.HS;
	double *iv  = ds.V;
	
	CompartmentStruct *ic;
	for ( 	ic = compartment;
			ic < compartment + nCompts * sizeof(CompartmentStruct);
			++ic ) {
		*ihs         = *( 2 + ihs );
		*( 3 + ihs ) = *iv * ic->CmByDt + ic->EmByRm;
		
		ihs += 4, ++iv;
	}
	
	/* Not going to consider inject at the first implementation level
	map< unsigned int, InjectStruct >::iterator inject;
	for ( inject = inject_.begin(); inject != inject_.end(); inject++ ) {
		unsigned int ic = inject->first;
		InjectStruct& value = inject->second;
		
		HS_[ 4 * ic + 3 ] += value.injectVarying + value.injectBasal;
		
		value.injectVarying = 0.0;
	}
	*/
}

void HSolvePassive::forwardEliminate(GpuDataStruct ds) {
	unsigned int ic = 0;
	double *ihs = ds.HS;
	OperandStruct *iop = ds.operand;
	JunctionStruct *junction;
	
	double pivot;
	double division;
	unsigned int index;
	unsigned int rank;
	for ( junction = ds.junction;
	      junction != ds.junction + ds.junctionSize;
	      junction++ )
	{
		index = junction->index;
		rank = junction->rank;
		
		while ( ic < index ) {
			*( ihs + 4 ) -= *( ihs + 1 ) / *ihs * *( ihs + 1 );
			*( ihs + 7 ) -= *( ihs + 1 ) / *ihs * *( ihs + 3 );
			
			++ic, ihs += 4;
		}
		
		pivot = *ihs;
		if ( rank == 1 ) {
			double *j = iop->ops;
			double *s = iop->ops + 1;
			
			division    = *( j + 1 ) / pivot;
			*( s )     -= division * *j;
			*( s + 3 ) -= division * *( ihs + 3 );
			
			iop += 3;
		} else if ( rank == 2 ) {
			vdIterator j = *iop;
			vdIterator s;
			
			s           = *( iop + 1 );
			division    = *( j + 1 ) / pivot;
			*( s )     -= division * *j;
			*( j + 4 ) -= division * *( j + 2 );
			*( s + 3 ) -= division * *( ihs + 3 );
			
			s           = *( iop + 3 );
			division    = *( j + 3 ) / pivot;
			*( j + 5 ) -= division * *j;
			*( s )     -= division * *( j + 2 );
			*( s + 3 ) -= division * *( ihs + 3 );
			
			iop += 5;
		} else {
			vector< vdIterator >::iterator
				end = iop + 3 * rank * ( rank + 1 );
			for ( ; iop < end; iop += 3 )
				**iop -= **( iop + 2 ) / pivot * **( iop + 1 );
		}
		
		++ic, ihs += 4;
	}
	
	while ( ic < nCompt_ - 1 ) {
		*( ihs + 4 ) -= *( ihs + 1 ) / *ihs * *( ihs + 1 );
		*( ihs + 7 ) -= *( ihs + 1 ) / *ihs * *( ihs + 3 );
		
		++ic, ihs += 4;
	}
}

void HSolvePassive::backwardSubstitute() {
	int ic = nCompt_ - 1;
	vector< double >::reverse_iterator ivmid = VMid_.rbegin();
	vector< double >::reverse_iterator iv = V_.rbegin();
	vector< double >::reverse_iterator ihs = HS_.rbegin();
	vector< vdIterator >::reverse_iterator iop = operand_.rbegin();
	vector< vdIterator >::reverse_iterator ibop = backOperand_.rbegin();
	vector< JunctionStruct >::reverse_iterator junction;
	
	*ivmid = *ihs / *( ihs + 3 );
	*iv = 2 * *ivmid - *iv;
	--ic, ++ivmid, ++iv, ihs += 4;
	
	int index;
	int rank;
	for ( junction = junction_.rbegin();
	      junction != junction_.rend();
	      junction++ )
	{
		index = junction->index;
		rank = junction->rank;
		
		while ( ic > index ) {
			*ivmid = ( *ihs - *( ihs + 2 ) * *( ivmid - 1 ) ) / *( ihs + 3 );
			*iv = 2 * *ivmid - *iv;
			
			--ic, ++ivmid, ++iv, ihs += 4;
		}
		
		if ( rank == 1 ) {
			*ivmid = ( *ihs - **iop * **( iop + 2 ) ) / *( ihs + 3 );
			
			iop += 3;
		} else if ( rank == 2 ) {
			vdIterator v0 = *( iop );
			vdIterator v1 = *( iop + 2 );
			vdIterator j  = *( iop + 4 );
			
			*ivmid = ( *ihs
			           - *v0 * *( j + 2 )
			           - *v1 * *j
			         ) / *( ihs + 3 );
			
			iop += 5;
		} else {
			*ivmid = *ihs;
			for ( int i = 0; i < rank; ++i ) {
				*ivmid -= **ibop * **( ibop + 1 );
				ibop += 2;
			}
			*ivmid /= *( ihs + 3 );
			
			iop += 3 * rank * ( rank + 1 );
		}
		
		*iv = 2 * *ivmid - *iv;
		--ic, ++ivmid, ++iv, ihs += 4;
	}
	
	while ( ic >= 0 ) {
		*ivmid = ( *ihs - *( ihs + 2 ) * *( ivmid - 1 ) ) / *( ihs + 3 );
		*iv = 2 * *ivmid - *iv;
		
		--ic, ++ivmid, ++iv, ihs += 4;
	}
}

