/**********************************************************************
** This program is part of 'MOOSE', the
** Messaging Object Oriented Simulation Environment.
**   copyright (C) 2003-2007 Upinder S. Bhalla, Niraj Dudani and NCBS
** It is made available under the terms of the
** GNU Lesser General Public License version 2.1
** See the file COPYING.LIB for the full notice.
**********************************************************************/

#include <vector>
#include "GpuInterface.h"
#include "GpuKernels.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/*
 * Check CUDA return value and handle appropriately
 */
#define _(value) {															\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		std::cerr << "Error " << hipGetErrorString(_m_cudaStat)			\
				  << " at line " << __LINE__ << " in file " << __FILE__		\
				  << std::endl;												\
		exit(1);															\
	} 																		\
}

/**
 * Constructor for the GpuInterface class.
 * Allocates memory for all data elements in the GPU. Transfers data from CPU
 * to GPU.
 */
GpuInterface::GpuInterface(HSolve *hsolve)
{
	// Save the pointer!
	hsolve_ = hsolve;

	// Find the required sizes of elements
	data_.nCompts = hsolve->V_.size();
	data_.HJSize = hsolve->HJ_.size();
	data_.junctionSize = hsolve->junction_.size();

	// Allocate memory for data members
	if ( data_.nCompts > 0 ) {
		_( hipMalloc((void **) &data_.HS, 4* data_.nCompts* sizeof(double)) );
		_( hipMalloc((void **) &data_.V, data_.nCompts * sizeof(double)) );
		_( hipMalloc((void **) &data_.VMid, data_.nCompts * sizeof(double)) );
		_( hipMalloc((void **) &data_.compartment,
					  data_.nCompts * sizeof(CompartmentStruct)) );
	} else {
		data_.HS = NULL;
		data_.V = NULL;
		data_.VMid = NULL;
		data_.compartment = NULL;
	}
	if ( data_.HJSize > 0 ) {
		_( hipMalloc((void **) &data_.HJ, data_.HJSize * sizeof(double)) );
		_( hipMalloc((void **) &data_.HJCopy, data_.HJSize* sizeof(double)) );
	} else {
		data_.HJ = NULL;
		data_.HJCopy = NULL;
	}
	if ( data_.junctionSize > 0 ) {
		_( hipMalloc( (void **) &data_.junction,
					   data_.junctionSize * sizeof(JunctionStruct) ) );
	} else {
		data_.junction = NULL;
	}

	// Copy array-of-double data into GPU
	if ( data_.HS )
		_( hipMemcpy( data_.HS, &hsolve->HS_[0],
					   4 * data_.nCompts * sizeof(double),
					   hipMemcpyHostToDevice ) );
	if ( data_.V )
		_( hipMemcpy( data_.V, &hsolve->V_[0],
					   data_.nCompts * sizeof(double),
					   hipMemcpyHostToDevice ) );
	if ( data_.VMid )
		_( hipMemcpy( data_.VMid, &hsolve->VMid_[0],
					   data_.nCompts * sizeof(double),
					   hipMemcpyHostToDevice ) );
	if ( data_.HJ )
		_( hipMemcpy( data_.HJ, &hsolve->HJ_[0],
					   data_.HJSize * sizeof(double),
					   hipMemcpyHostToDevice ) );
	if ( data_.HJCopy )
		_( hipMemcpy( data_.HJCopy, &hsolve->HJCopy_[0],
					   data_.HJSize * sizeof(double),
					   hipMemcpyHostToDevice ) );
	// Copy data for array-of-struct data members
	if ( data_.compartment )
		_( hipMemcpy( data_.compartment, &hsolve->compartment_[0],
					   data_.nCompts * sizeof(CompartmentStruct),
					   hipMemcpyHostToDevice ) );
	if ( data_.junction )
		_( hipMemcpy( data_.junction, &hsolve->junction_[0],
					   data_.junctionSize * sizeof(JunctionStruct),
					   hipMemcpyHostToDevice ) );

	// Call to take care of populating GpuInterface::operand_ and
	// GpuInterface::backOperand_.
	makeOperands(hsolve);

	// Allocate and copy memory for operands and backOperands
	data_.operandSize = operand_.size();
	data_.backOperandSize = backOperand_.size();

	if ( data_.operandSize > 0 ) {
		_( hipMalloc( (void**)&data_.operand,
					   operand_.size() * sizeof(double*) ) );
		_( hipMemcpy( data_.operand, &operand_[ 0 ],
					   operand_.size() * sizeof(double*),
					   hipMemcpyHostToDevice ) );
	} else {
		data_.operand = NULL;
	}

	if ( data_.backOperandSize > 0 ) {
		_( hipMalloc((void**)&data_.backOperand,
					  backOperand_.size() * sizeof(double*)) );
		_( hipMemcpy(data_.backOperand, &backOperand_[ 0 ],
					  backOperand_.size() * sizeof(double*),
					  hipMemcpyHostToDevice) );
	} else {
		data_.backOperand = NULL;
	}

#ifdef DO_UNIT_TESTS
	if ( data_.nCompts > 0 ) {
		_( hipMalloc( (void **) &data_.inject,
					   data_.nCompts * sizeof(InjectStruct) ) );
		copyInject();
	} else {
		data_.inject = NULL;
	}
#endif

	// Need to decide how many blocks and threads to use per HSolve object
	// For now, keep each hsolver on its own thread.
	numBlocks_ = 1;
	numThreads_ = 1;
}

/**
 * Function to take care of making operands in the same way that
 * HinesMatrix::makeOperands does.
 */
void GpuInterface::makeOperands(HSolve *hsolve)
{
	typedef vector< double >::iterator vdIterator;

	unsigned int index;
	unsigned int rank;
	unsigned int farIndex;
	double *base;
	vector< JunctionStruct >::iterator junction;
	
	// Operands for forward-elimination
	for ( junction = hsolve->junction_.begin();
		  junction != hsolve->junction_.end();
		  ++junction )
	{
		index = junction->index;
		rank = junction->rank;

		// operandBase_[ index ] maps to the vdIterator corresponding to the
		// position of compartment with Hines index `index` in HJ_.
		// base needs to contain the pointer to HJ (in the GPU) which marks
		// the start of this juction in HJ.
		base = data_.HJ
			   + (long)( &( *hsolve->operandBase_[index] ) - &hsolve->HJ_[0] );

		// This is the list of compartments connected at a junction.
		const vector< unsigned int >& group =
			hsolve->coupled_[ hsolve->groupNumber_[ index ] ];
		
		if ( rank == 1 ) {
			operand_.push_back( base );
			
			// Select last member.
			farIndex = group[ group.size() - 1 ];
			operand_.push_back( data_.HS + 4 * farIndex );
			operand_.push_back( data_.VMid + farIndex );
		} else if ( rank == 2 ) {
			operand_.push_back( base );
			
			// Select 2nd last member.
			farIndex = group[ group.size() - 2 ];
			operand_.push_back( data_.HS + 4 * farIndex );
			operand_.push_back( data_.VMid + farIndex );
			
			// Select last member.
			farIndex = group[ group.size() - 1 ];
			operand_.push_back( data_.HS + 4 * farIndex );
			operand_.push_back( data_.VMid + farIndex );
		} else {
			// Operations on diagonal elements and elements from B
			// (as in Ax = B).
			int start = group.size() - rank;
			for ( unsigned int j = 0; j < rank; ++j ) {
				farIndex = group[ start + j ];
				
				// Diagonal elements
				operand_.push_back( data_.HS + 4 * farIndex );
				operand_.push_back( base + 2 * j );
				operand_.push_back( base + 2 * j + 1 );
				
				// Elements from B
				operand_.push_back( data_.HS + 4 * farIndex + 3 );
				operand_.push_back( data_.HS + 4 * index + 3 );
				operand_.push_back( base + 2 * j + 1 );
			}
			
			// Operations on off-diagonal elements.
			double *left;
			double *above;
			double *target;
			
			// Upper triangle elements
			left = base + 1;
			target = base + 2 * rank;
			for ( unsigned int i = 1; i < rank; ++i ) {
				above = base + 2 * i;
				for ( unsigned int j = 0; j < rank - i; ++j ) {
					operand_.push_back( target );
					operand_.push_back( above );
					operand_.push_back( left );
					
					above += 2;
					target += 2;
				}
				left += 2;
			}
			
			// Lower triangle elements
			target = base + 2 * rank + 1;
			above = base;
			for ( unsigned int i = 1; i < rank; ++i ) {
				left = base + 2 * i + 1;
				for ( unsigned int j = 0; j < rank - i; ++j ) {
					operand_.push_back( target );
					operand_.push_back( above );
					operand_.push_back( left );
					
					/*
					 * This check required because the MS VC++ compiler is
					 * paranoid about iterators going out of bounds, even if
					 * they are never used after that.
					 */
					if ( i == rank - 1 && j == rank - i - 1 )
						continue;
					
					target += 2;
					left += 2;
				}
				above += 2;
			}
		}
	}
	
	// Operands for backward substitution
	for ( junction = hsolve->junction_.begin();
		  junction != hsolve->junction_.end();
		  ++junction )
	{
		if ( junction->rank < 3 )
			continue;
		
		index = junction->index;
		rank = junction->rank;
		base = data_.HJ
			   + (long)( &( *hsolve->operandBase_[index] ) - &hsolve->HJ_[0] );
		
		// This is the list of compartments connected at a junction.
		const vector< unsigned int >& group =
			hsolve->coupled_[ hsolve->groupNumber_[ index ] ];
		
		unsigned int start = group.size() - rank;
		for ( unsigned int j = 0; j < rank; ++j ) {
			farIndex = group[ start + j ];
			
			backOperand_.push_back( base + 2 * j );
			backOperand_.push_back( data_.VMid + farIndex );
		}
	}
}

void GpuInterface::gpuUpdateMatrix()
{
	dim3 numBlocks(numBlocks_);
	dim3 numThreads(numThreads_);

	updateMatrixKernel<<< numBlocks, numThreads >>>( data_ );

	stage_ = 0;    // Update done.
}

void GpuInterface::gpuForwardEliminate()
{
	dim3 numBlocks(numBlocks_);
	dim3 numThreads(numThreads_);

	forwardEliminateKernel<<< numBlocks, numThreads >>>( data_ );

	stage_ = 1;    // Forward elimination done.
}

void GpuInterface::gpuBackwardSubstitute()
{
	dim3 numBlocks(numBlocks_);
	dim3 numThreads(numThreads_);

	backwardSubstituteKernel<<< numBlocks, numThreads >>>( data_ );

	stage_ = 2;    // Backward substitution done.
}

void GpuInterface::synchronize()
{
	hipDeviceSynchronize();
}

void GpuInterface::unsetup()
{
	// Create temporary storage space before assigning the vectors in HSolve.
	double *HS = new double[ 4 * data_.nCompts ];
	double *HJ = new double[ data_.HJSize ];
	double *V = new double[ data_.nCompts ];
	double **operand = new double*[ data_.operandSize ];

	// Copy data from the GPU back to the CPU and then into the HSolve vectors
	_( hipMemcpy( HS, data_.HS, 4 * data_.nCompts * sizeof(double),
				   hipMemcpyDeviceToHost ) );
	hsolve_->HS_.assign( HS, HS + 4 * data_.nCompts );

	_( hipMemcpy( HJ, data_.HJ, data_.HJSize * sizeof(double),
				   hipMemcpyDeviceToHost ) );
	hsolve_->HJ_.assign( HJ, HJ + data_.HJSize );

	_( hipMemcpy( HJ, data_.HJCopy, data_.HJSize * sizeof(double),
				   hipMemcpyDeviceToHost ) );
	hsolve_->HJCopy_.assign( HJ, HJ + data_.HJSize );

	_( hipMemcpy( V, data_.V, data_.nCompts * sizeof(double),
				   hipMemcpyDeviceToHost ) );
	hsolve_->V_.assign( V, V + data_.nCompts );

	_( hipMemcpy( V, data_.VMid, data_.nCompts * sizeof(double),
				   hipMemcpyDeviceToHost ) );
	hsolve_->VMid_.assign( V, V + data_.nCompts );
	
	_( hipMemcpy( operand, data_.operand, data_.operandSize * sizeof(double),
				   hipMemcpyDeviceToHost ) );
	operand_.assign( operand, operand + data_.operandSize );

}

#ifdef DO_UNIT_TESTS

/**
 * Function to copy inject_ from the CPU to the GPU. This is used only for
 * testing the RC-behaviour of a single compartment. Hence it is being defined
 * only if unit tests are performed
 */
void GpuInterface::copyInject()
{
	map< unsigned int, InjectStruct >::iterator i;
	vector< InjectStruct > inject( data_.nCompts, InjectStruct() );

	for ( i = hsolve_->inject_.begin(); i != hsolve_->inject_.end(); ++i ) {
		unsigned int ic = i->first;
		InjectStruct& value = i->second;
		inject[ ic ] = value;
	}

	// Memory must already be allocated. This should have happened during
	// construction of the object.
	_( hipMemcpy( data_.inject, &inject[ 0 ],
				   data_.nCompts * sizeof(InjectStruct),
				   hipMemcpyHostToDevice ) );
}

// getA and getB functions used in unit tests for comparing matrix element
// values.

/**
 * Used by getA and getB to retrieve single data elements from the GPU.
 * Horribly inefficient.
 */
template< class T >
T get(T *address) {
	T value;
	// Copy data from GPU to CPU
	_( hipMemcpy( &value, address, sizeof( T ), hipMemcpyDeviceToHost ) );
	return value;
}
#define getd( addr ) get< double >( addr )

/**
 * Get the (row, col)-element of the Hines matrix.
 */
double GpuInterface::getA( unsigned int row, unsigned int col ) const
{
	/*
	 * If forward elimination is done, or backward substitution is done, and
	 * if (row, col) is in the lower triangle, then return 0.
	 */
	if ( ( stage_ == 1 || stage_ == 2 ) && row > col )
		return 0.0;

	if ( row >= data_.nCompts || col >= data_.nCompts )
		return 0.0;

	if ( row == col ) {
		return getd( data_.HS + 4 * row );
	}

	unsigned int smaller = row < col ? row : col;
	unsigned int bigger = row > col ? row : col;

	// If find returns end, it means that `smaller` was not found.
	if ( hsolve_->groupNumber_.find(smaller) == hsolve_->groupNumber_.end() ) {
		if ( bigger - smaller == 1 )
			return getd( data_.HS + 4 * smaller + 1 );
		else
			return 0.0;
	} else {
		// We could use: groupNumber = groupNumber_[ smaller ], but this is a
		// const function
		unsigned int groupNumber = hsolve_->groupNumber_.find(smaller)->second;
		const vector< unsigned int >& group = hsolve_->coupled_[ groupNumber ];
		unsigned int location, size;
		unsigned int smallRank, bigRank;

		if ( find( group.begin(), group.end(), bigger ) != group.end() ) {
			location = 0;
			for ( int i = 0; i < static_cast< int >( groupNumber ); ++i ) {
				size = hsolve_->coupled_[ i ].size();
				location += size * ( size - 1 );
			}

			size = group.size();
			smallRank = group.end()
						- find( group.begin(), group.end(), smaller ) - 1;
			bigRank = group.end()
					  - find( group.begin(), group.end(), bigger ) - 1;
			location += size * ( size - 1 ) - smallRank * ( smallRank + 1 );
			location += 2 * ( smallRank - bigRank - 1 );

			if ( row == smaller )
				return getd( data_.HJ + location );
			else
				return getd( data_.HJ + location + 1 );
		} else {
			return 0.0;
		}
	}
}

double GpuInterface::getB( unsigned int row ) const
{
	return getd( data_.HS + 4 * row + 3 );
}

double GpuInterface::getVMid( unsigned int row ) const
{
	return getd( data_.VMid + row );
}

double GpuInterface::getV( unsigned int row ) const
{
	return getd( data_.V + row );
}

#endif // DO_UNIT_TESTS

