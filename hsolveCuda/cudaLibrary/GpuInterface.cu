/**********************************************************************
** This program is part of 'MOOSE', the
** Messaging Object Oriented Simulation Environment.
**   copyright (C) 2003-2007 Upinder S. Bhalla, Niraj Dudani and NCBS
** It is made available under the terms of the
** GNU Lesser General Public License version 2.1
** See the file COPYING.LIB for the full notice.
**********************************************************************/

#include "GpuInterface.h"
#include "GpuKernels.h"

/*
 * Check CUDA return value and handle appropriately
 */
#define _(value) {															\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

/* 
 * Constructor for the GpuInterface class.
 * Allocates memory for all data elements in the GPU
 * Transfers data from CPU to GPU.
 */
GpuInterface::GpuInterface(HSolve *hsolve)
{
	// Find the required sizes of elements
	data_.nCompts = hsolve->V_.size();
	data_.HJSize = hsolve->HJ_.size();
	data_.operandSize = hsolve->operand_.size();
	data_.backOperandSize = hsolve->backOperand_.size();
	data_.junctionSize = hsolve->junction_.size();
	
	// Allocate memory for array-of-double data members
	_( hipMalloc( (void **) &data_.HS, 4 * data_.nCompts * sizeof(double) ) );
	_( hipMalloc( (void **) &data_.HJ, data_.HJSize * sizeof(double) ) );
	_( hipMalloc( (void **) &data_.V, data_.nCompts * sizeof(double) ) );
	_( hipMalloc( (void **) &data_.VMid, data_.nCompts * sizeof(double) ) );
	_( hipMalloc( (void **) &data_.HJCopy, data_.HJSize * sizeof(double) ) );
	
	// Copy array-of-double data into GPU
	_( hipMemcpy( data_.HS, &hsolve->HS_[0], 4* data_.nCompts* sizeof(double),
				   hipMemcpyHostToDevice ) );
	_( hipMemcpy( data_.HJ, &hsolve->HJ_[0], data_.HJSize * sizeof(double),
				   hipMemcpyHostToDevice ) );
	_( hipMemcpy( data_.V, &hsolve->V_[0], data_.nCompts * sizeof(double),
				   hipMemcpyHostToDevice ) );
	_( hipMemcpy( data_.VMid, &hsolve->VMid_[0], data_.nCompts*sizeof(double),
				   hipMemcpyHostToDevice ) );
	_( hipMemcpy( data_.HJCopy, &hsolve->HJCopy_[0],
				   data_.HJSize * sizeof(double), hipMemcpyHostToDevice ) );

	// Allocate memory for array-of-struct data members that do not contain
	// pointers
	_( hipMalloc( (void **) &data_.compartment,
				   data_.nCompts * sizeof(Compartment) ) );
	_( hipMalloc( (void **) &data_.junction,
				   data_.junctionSize * sizeof(Junction) ) );
	
	// Copy data for array-of-struct data members that do not contain pointers
	_( hipMalloc( data_.compartment, &hsolve->compartment_[0],
				   data_.nCompts * sizeof(Compartment),
				   hipMemcpyHostToDevice ) );
	_( hipMalloc( data_.junction, &hsolve->junction_[0],
				   data_.junctionSize * sizeof(Junction),
				   hipMemcpyHostToDevice ) );
	
	// Allocate data for array-of-struct data members that contain pointers

	// First, we need to create the structs out of the vector of vectors.
	OperandStruct *os = new OperandStruct[data_.operandSize];
	for( int i = 0 ; i < data_.operandSize ; i++ )
	{
		// Find the number of operands in the ith vector of hsolve->operand_
		os[i].nOps = hsolve->operand_[i].size();
		// Allocate memory for the ith vector in hsolve->operand_
		_( hipMalloc( (void **) &os[i].ops, os[i].nOps * sizeof(double) ) );
		// Copy data for the ith vector in hsolve->operand
		_( hipMemcpy( &os[i].ops, &hsolve->operand_[i][0],
					   os[i].nOps * sizeof(double), hipMemcpyHostToDevice ) );
	}
	// Finally, copy the entire set of pointers to these operand arrays into
	// the GPU
	_( hipMalloc( (void **) &data_.operand,
				   data_.operandSize * sizeof(OperandStruct) ) );
	_( hipMemcpy( data_.operand, os,
				   data_.operandSize * sizeof(OperandStruct),
				   hipMemcpyHostToDevice ) );

	// Now, to do the same for hsolve->backOperand_
	OperandStruct *bos = new OperandStruct[data_.backOperandSize];
	for( int i = 0 ; i < data_.backOperandSize ; i++ )
	{
		// Find the number of operands in the ith vector of hsolve->operand_
		bos[i].nOps = hsolve->backOperand_[i].size();
		// Allocate memory for the ith vector in hsolve->operand_
		_( hipMalloc( (void **) &bos[i].ops, bos[i].nOps * sizeof(double) ) );
		// Copy data for the ith vector in hsolve->operand
		_( hipMemcpy( &bos[i].ops, &hsolve->backOperand_[i][0],
					   bos[i].nOps * sizeof(double), hipMemcpyHostToDevice) );
	}
	// Finally, copy the entire set of pointers to these operand arrays into
	// the GPU
	_( hipMalloc( (void **) &data_.backOperand,
				   data_.backOperandSize * sizeof(OperandStruct) ) );
	_( hipMemcpy( data_.backOperand, bos,
				   data_.backOperandSize * sizeof(OperandStruct),
				   hipMemcpyHostToDevice ) );

	// Need to decide how many blocks and threads to use per HSolve object
	// For now, keep each hsolver on its own thread.
	numBlocks_ = 1;
	numThreads_ = 1;
}

void GpuInterface::gpuUpdateMatrix()
{
	dim3 numBlocks(numBlocks_);
	dim3 numThreads(numThreads_);

	gpuUpdateMatrix<<< numBlocks, numThreads >>>( data_ );

	stage_ = 0;    // Update done.
}

void GpuInterface::gpuForwardEliminate()
{
	dim3 numBlocks(numBlocks_);
	dim3 numThreads(numThreads_);

	gpuForwardEliminate<<< numBlocks, numThreads >>>( data_ );

	stage_ = 1;    // Forward elimination done.
}

void GpuInterface::gpuBackwardSubstitute()
{
	dim3 numBlocks(numBlocks_);
	dim3 numThreads(numThreads_);

	gpuBackwardSubstitute<<< numBlocks, numThreads >>>( data_ );
	
	stage_ = 2;    // Backward substitution done.
}
