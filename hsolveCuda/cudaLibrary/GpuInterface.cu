/**********************************************************************
** This program is part of 'MOOSE', the
** Messaging Object Oriented Simulation Environment.
**   copyright (C) 2003-2007 Upinder S. Bhalla, Niraj Dudani and NCBS
** It is made available under the terms of the
** GNU Lesser General Public License version 2.1
** See the file COPYING.LIB for the full notice.
**********************************************************************/

#include <vector>
#include "GpuInterface.h"
#include "GpuKernels.h"

/*
 * Check CUDA return value and handle appropriately
 */
#define _(value) {															\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		std::cerr << "Error " << hipGetErrorString(_m_cudaStat)			\
				  << " at line " << __LINE__ << " in file " << __FILE__		\
				  << std::endl;												\
		exit(1);															\
	} }

/** 
 * Constructor for the GpuInterface class.
 * Allocates memory for all data elements in the GPU. Transfers data from CPU
 * to GPU.
 */
GpuInterface::GpuInterface(HSolve *hsolve)
{
	// Find the required sizes of elements
	data_.nCompts = hsolve->V_.size();
	data_.HJSize = hsolve->HJ_.size();
	data_.junctionSize = hsolve->junction_.size();

	// Allocate memory for array-of-double data members
	_( hipMalloc( (void **) &data_.HS, 4 * data_.nCompts * sizeof(double) ) );
	_( hipMalloc( (void **) &data_.HJ, data_.HJSize * sizeof(double) ) );
	_( hipMalloc( (void **) &data_.V, data_.nCompts * sizeof(double) ) );
	_( hipMalloc( (void **) &data_.VMid, data_.nCompts * sizeof(double) ) );
	_( hipMalloc( (void **) &data_.HJCopy, data_.HJSize * sizeof(double) ) );

	// Copy array-of-double data into GPU
	_( hipMemcpy( data_.HS, &hsolve->HS_[0], 4* data_.nCompts* sizeof(double),
				   hipMemcpyHostToDevice ) );
	_( hipMemcpy( data_.HJ, &hsolve->HJ_[0], data_.HJSize * sizeof(double),
				   hipMemcpyHostToDevice ) );
	_( hipMemcpy( data_.V, &hsolve->V_[0], data_.nCompts * sizeof(double),
				   hipMemcpyHostToDevice ) );
	_( hipMemcpy( data_.VMid, &hsolve->VMid_[0], data_.nCompts*sizeof(double),
				   hipMemcpyHostToDevice ) );
	_( hipMemcpy( data_.HJCopy, &hsolve->HJCopy_[0],
				   data_.HJSize * sizeof(double), hipMemcpyHostToDevice ) );

	// Allocate memory for array-of-structure data members
	_( hipMalloc( (void **) &data_.compartment,
				   data_.nCompts * sizeof(CompartmentStruct) ) );
	_( hipMalloc( (void **) &data_.junction,
				   data_.junctionSize * sizeof(JunctionStruct) ) );

	// Copy data for array-of-struct data members
	_( hipMemcpy( data_.compartment, &hsolve->compartment_[0],
				   data_.nCompts * sizeof(CompartmentStruct),
				   hipMemcpyHostToDevice ) );
	_( hipMemcpy( data_.junction, &hsolve->junction_[0],
				   data_.junctionSize * sizeof(JunctionStruct),
				   hipMemcpyHostToDevice ) );

	// Call to take care of populating GpuInterface::operand_
	makeOperands(hsolve);

	// Allocate and copy memory for operands and backOperands
	data_.operandSize = operand_.size();
	data_.backOperandSize = backOperand_.size();

	_( hipMalloc((void**)&data_.operand, operand_.size() * sizeof(double*)) );
	_( hipMemcpy(data_.operand, &operand_[ 0 ],
				  operand_.size() * sizeof(double*), hipMemcpyHostToDevice) );

	_( hipMalloc((void**)&data_.backOperand,
				  backOperand_.size() * sizeof(double*)) );
	_( hipMemcpy(data_.backOperand, &backOperand_[ 0 ],
				  backOperand_.size() * sizeof(double*),
				  hipMemcpyHostToDevice) );

	// Need to decide how many blocks and threads to use per HSolve object
	// For now, keep each hsolver on its own thread.
	numBlocks_ = 1;
	numThreads_ = 1;
}

/**
 * Function to take care of making operands in the same way that
 * HinesMatrix::makeOperands does.
 */
void GpuInterface::makeOperands(HSolve *hsolve)
{
	typedef vector< double >::iterator vdIterator;

	unsigned int index;
	unsigned int rank;
	unsigned int farIndex;
	double *base;
	vector< JunctionStruct >::iterator junction;
	
	// Operands for forward-elimination
	for ( junction = hsolve->junction_.begin();
		  junction != hsolve->junction_.end();
		  ++junction )
	{
		index = junction->index;
		rank = junction->rank;

		// operandBase_[ index ] maps to the vdIterator corresponding to the
		// position of compartment with Hines index `index` in HJ_.
		// base needs to contain the pointer to HJ (in the GPU) which marks
		// the start of this juction in HJ.
		base = data_.HJ
			   + (long)( &( *hsolve->operandBase_[index] ) - &hsolve->HJ_[0] );

		// This is the list of compartments connected at a junction.
		const vector< unsigned int >& group =
			hsolve->coupled_[ hsolve->groupNumber_[ index ] ];
		
		if ( rank == 1 ) {
			operand_.push_back( base );
			
			// Select last member.
			farIndex = group[ group.size() - 1 ];
			operand_.push_back( &data_.HS[ 0 ] + 4 * farIndex );
			operand_.push_back( &data_.VMid[ 0 ] + farIndex );
		} else if ( rank == 2 ) {
			operand_.push_back( base );
			
			// Select 2nd last member.
			farIndex = group[ group.size() - 2 ];
			operand_.push_back( &data_.HS[ 0 ] + 4 * farIndex );
			operand_.push_back( &data_.VMid[ 0 ] + farIndex );
			
			// Select last member.
			farIndex = group[ group.size() - 1 ];
			operand_.push_back( &data_.HS[ 0 ] + 4 * farIndex );
			operand_.push_back( &data_.VMid[ 0 ] + farIndex );
		} else {
			// Operations on diagonal elements and elements from B
			// (as in Ax = B).
			int start = group.size() - rank;
			for ( unsigned int j = 0; j < rank; ++j ) {
				farIndex = group[ start + j ];
				
				// Diagonal elements
				operand_.push_back( &data_.HS [ 0 ] + 4 * farIndex );
				operand_.push_back( base + 2 * j );
				operand_.push_back( base + 2 * j + 1 );
				
				// Elements from B
				operand_.push_back( &data_.HS[ 0 ] + 4 * farIndex + 3 );
				operand_.push_back( &data_.HS[ 0 ] + 4 * index + 3 );
				operand_.push_back( base + 2 * j + 1 );
			}
			
			// Operations on off-diagonal elements.
			double *left;
			double *above;
			double *target;
			
			// Upper triangle elements
			left = base + 1;
			target = base + 2 * rank;
			for ( unsigned int i = 1; i < rank; ++i ) {
				above = base + 2 * i;
				for ( unsigned int j = 0; j < rank - i; ++j ) {
					operand_.push_back( target );
					operand_.push_back( above );
					operand_.push_back( left );
					
					above += 2;
					target += 2;
				}
				left += 2;
			}
			
			// Lower triangle elements
			target = base + 2 * rank + 1;
			above = base;
			for ( unsigned int i = 1; i < rank; ++i ) {
				left = base + 2 * i + 1;
				for ( unsigned int j = 0; j < rank - i; ++j ) {
					operand_.push_back( target );
					operand_.push_back( above );
					operand_.push_back( left );
					
					/*
					 * This check required because the MS VC++ compiler is
					 * paranoid about iterators going out of bounds, even if
					 * they are never used after that.
					 */
					if ( i == rank - 1 && j == rank - i - 1 )
						continue;
					
					target += 2;
					left += 2;
				}
				above += 2;
			}
		}
	}
	
	// Operands for backward substitution
	for ( junction = hsolve->junction_.begin();
		  junction != hsolve->junction_.end();
		  ++junction )
	{
		if ( junction->rank < 3 )
			continue;
		
		index = junction->index;
		rank = junction->rank;
		base = data_.HJ
			   + (long)( &( *hsolve->operandBase_[index] ) - &hsolve->HJ_[0] );
		
		// This is the list of compartments connected at a junction.
		const vector< unsigned int >& group =
			hsolve->coupled_[ hsolve->groupNumber_[ index ] ];
		
		unsigned int start = group.size() - rank;
		for ( unsigned int j = 0; j < rank; ++j ) {
			farIndex = group[ start + j ];
			
			backOperand_.push_back( base + 2 * j );
			backOperand_.push_back( &data_.VMid[ 0 ] + farIndex );
		}
	}
}

void GpuInterface::gpuUpdateMatrix()
{
	dim3 numBlocks(numBlocks_);
	dim3 numThreads(numThreads_);

	updateMatrixKernel<<< numBlocks, numThreads >>>( data_ );

	stage_ = 0;    // Update done.
}

void GpuInterface::gpuForwardEliminate()
{
	dim3 numBlocks(numBlocks_);
	dim3 numThreads(numThreads_);

	forwardEliminateKernel<<< numBlocks, numThreads >>>( data_ );

	stage_ = 1;    // Forward elimination done.
}

void GpuInterface::gpuBackwardSubstitute()
{
	dim3 numBlocks(numBlocks_);
	dim3 numThreads(numThreads_);

	backwardSubstituteKernel<<< numBlocks, numThreads >>>( data_ );
	
	stage_ = 2;    // Backward substitution done.
}


